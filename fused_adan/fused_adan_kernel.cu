#include "hip/hip_runtime.h"
/* Copyright 2021 The LightSeq Team
   Copyright NVIDIA/apex
   Copyright AlexwellChen
   This kernel is adapted from NVIDIA/apex and LightSeq Team
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cmath>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/Exceptions.h>
#include "include/type_shim.h"
#include "include/fused_adan_kernel.cuh"


template <typename T, typename GRAD_T>
__global__ void adan_cuda_kernel(
    T* __restrict__ p,
    GRAD_T* __restrict__ p_copy,  // For mixed precision training, pass NULL if
                                  // not needed
    const GRAD_T* __restrict__ g, T* __restrict__ exp_avg, T* __restrict__ exp_avg_sq, T* __restrict__ exp_avg_diff,
    const GRAD_T* __restrict__ neg_grad, const float b1, const float b2, const float b3, 
    const float bias_correction1, const float bias_correction2, const float bias_correction3_sqrt,
    const float lr, const float decay, const float eps, const bool no_prox, const float clip_global_grad_norm, const size_t total_size
    ){
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_id >= total_size) return;

    T scaled_grad = g[global_id] * clip_global_grad_norm;

    GRAD_T diff, update;

    diff = scaled_grad + neg_grad[global_id];
    update = scaled_grad + b2 * diff;

    exp_avg[global_id] = b1 * exp_avg[global_id] + (1 - b1) * scaled_grad;

    exp_avg_diff[global_id] = b2 * exp_avg_diff[global_id] + (1 - b2) * diff;

    exp_avg_sq[global_id] = b3 * exp_avg_sq[global_id] + (1 - b3) * update * update;

    float denom;
    denom = sqrtf(exp_avg_sq[global_id]) / bias_correction3_sqrt + eps;
    update = (exp_avg[global_id] / bias_correction1 + b2 * exp_avg_diff[global_id] / bias_correction2) / denom;
    
    if (no_prox){
        p[global_id] = p[global_id] * (1 - lr * decay) + update * (-lr);
    }else{
        p[global_id] = p[global_id] + update * (-lr) / (1 + lr * decay);
    } 
    if (p_copy != NULL) p_copy[global_id] = (GRAD_T)p[global_id];
}

template <>
__global__ void adan_cuda_kernel<float, float>(
    float* __restrict__ p,
    float* __restrict__ p_copy,  // For mixed precision training, pass NULL if
                                  // not needed
    const float* __restrict__ g, float* __restrict__ exp_avg, float* __restrict__ exp_avg_sq, float* __restrict__ exp_avg_diff,
    const float* __restrict__ neg_grad, const float b1, const float b2, const float b3, 
    const float bias_correction1, const float bias_correction2, const float bias_correction3_sqrt,
    const float lr, const float decay, const float eps, const bool no_prox, const float clip_global_grad_norm, const size_t total_size){

        int global_id = blockIdx.x * blockDim.x + threadIdx.x;

        if (global_id * 4 >= total_size) return;

        float4* p4_ptr = reinterpret_cast<float4*>(p);
        const float4* g4_ptr = reinterpret_cast<const float4*>(g);
        const float4* neg_grad4_diff_ptr = reinterpret_cast<const float4*>(neg_grad);
        float4* exp_avg4_ptr = reinterpret_cast<float4*>(exp_avg);
        float4* exp_avg_sq4_ptr = reinterpret_cast<float4*>(exp_avg_sq);
        float4* exp_avg_diff4_ptr = reinterpret_cast<float4*>(exp_avg_diff);
        
        float4 p4 = p4_ptr[global_id];
        const float4 g4 = g4_ptr[global_id];
        const float4 neg_grad4_diff = neg_grad4_diff_ptr[global_id];
        float4 exp_avg4 = exp_avg4_ptr[global_id];
        float4 exp_avg_sq4 = exp_avg_sq4_ptr[global_id];
        float4 exp_avg_diff4 = exp_avg_diff4_ptr[global_id];

        float4 new_p4;
        float4 new_exp_avg4;
        float4 new_exp_avg_sq4;
        float4 new_exp_avg_diff4;

        float scaled_grad1 = g4.x * clip_global_grad_norm;
        float scaled_grad2 = g4.y * clip_global_grad_norm;
        float scaled_grad3 = g4.z * clip_global_grad_norm;
        float scaled_grad4 = g4.w * clip_global_grad_norm;

        neg_grad4_diff.x = scaled_grad1 + neg_grad4_diff.x;
        neg_grad4_diff.y = scaled_grad2 + neg_grad4_diff.y;
        neg_grad4_diff.z = scaled_grad3 + neg_grad4_diff.z;
        neg_grad4_diff.w = scaled_grad4 + neg_grad4_diff.w;

        float update1 = scaled_grad1 + b2 * neg_grad4_diff.x;
        float update2 = scaled_grad2 + b2 * neg_grad4_diff.y;
        float update3 = scaled_grad3 + b2 * neg_grad4_diff.z;
        float update4 = scaled_grad4 + b2 * neg_grad4_diff.w;

        new_exp_avg4.x = b1 * exp_avg4.x + (1 - b1) * scaled_grad1;
        new_exp_avg4.y = b1 * exp_avg4.y + (1 - b1) * scaled_grad2;
        new_exp_avg4.z = b1 * exp_avg4.z + (1 - b1) * scaled_grad3;
        new_exp_avg4.w = b1 * exp_avg4.w + (1 - b1) * scaled_grad4;

        new_exp_avg_sq4.x = b3 * exp_avg_sq4.x + (1 - b3) * update1 * update1;
        new_exp_avg_sq4.y = b3 * exp_avg_sq4.y + (1 - b3) * update2 * update2;
        new_exp_avg_sq4.z = b3 * exp_avg_sq4.z + (1 - b3) * update3 * update3;
        new_exp_avg_sq4.w = b3 * exp_avg_sq4.w + (1 - b3) * update4 * update4;

        new_exp_avg_diff4.x = b2 * exp_avg_diff4.x + (1 - b2) * neg_grad4_diff.x;
        new_exp_avg_diff4.y = b2 * exp_avg_diff4.y + (1 - b2) * neg_grad4_diff.y;
        new_exp_avg_diff4.z = b2 * exp_avg_diff4.z + (1 - b2) * neg_grad4_diff.z;
        new_exp_avg_diff4.w = b2 * exp_avg_diff4.w + (1 - b2) * neg_grad4_diff.w;

        float4 denom4;
        denom4.x = sqrt(new_exp_avg_sq4.x - new_exp_avg_diff4.x * new_exp_avg_diff4.x / b2) + eps;
        denom4.y = sqrt(new_exp_avg_sq4.y - new_exp_avg_diff4.y * new_exp_avg_diff4.y / b2) + eps;
        denom4.z = sqrt(new_exp_avg_sq4.z - new_exp_avg_diff4.z * new_exp_avg_diff4.z / b2) + eps;
        denom4.w = sqrt(new_exp_avg_sq4.w - new_exp_avg_diff4.w * new_exp_avg_diff4.w / b2) + eps;

        // update = (exp_avg[global_id] / bias_correction1 + b2 * exp_avg_diff[global_id] / bias_correction2) / denom;
        update1 = (new_exp_avg4.x / bias_correction1 + b2 * new_exp_avg_diff4.x / bias_correction2) / denom4.x;
        update2 = (new_exp_avg4.y / bias_correction1 + b2 * new_exp_avg_diff4.y / bias_correction2) / denom4.y;
        update3 = (new_exp_avg4.z / bias_correction1 + b2 * new_exp_avg_diff4.z / bias_correction2) / denom4.z;
        update4 = (new_exp_avg4.w / bias_correction1 + b2 * new_exp_avg_diff4.w / bias_correction2) / denom4.w;

        if (no_prox){
            // p[global_id] = p[global_id] * (1 - lr * decay) + update * (-lr);
            new_p4.x = p4.x * (1 - lr * decay) + update1 * (-lr);
            new_p4.y = p4.y * (1 - lr * decay) + update2 * (-lr);
            new_p4.z = p4.z * (1 - lr * decay) + update3 * (-lr);
            new_p4.w = p4.w * (1 - lr * decay) + update4 * (-lr);
        }else{
            // p[global_id] = p[global_id] + update * (-lr) / (1 + lr * decay);
            new_p4.x = p4.x + update1 * (-lr) / (1 + lr * decay);
            new_p4.y = p4.y + update2 * (-lr) / (1 + lr * decay);
            new_p4.z = p4.z + update3 * (-lr) / (1 + lr * decay);
            new_p4.w = p4.w + update4 * (-lr) / (1 + lr * decay);
        }   

        p4_ptr[global_id] = new_p4;
        exp_avg4_ptr[global_id] = new_exp_avg4;
        exp_avg_sq4_ptr[global_id] = new_exp_avg_sq4;
        exp_avg_diff4_ptr[global_id] = new_exp_avg_diff4;
}

void fused_adan_cuda(at::Tensor& p, at::Tensor& p_copy, at::Tensor& g, at::Tensor& exp_avg, 
          at::Tensor& exp_avg_sq, at::Tensor& exp_avg_diff,
          at::Tensor& neg_grad, float beta1, float beta2, float beta3, 
          float bias_correction1, float bias_correction2, float bias_correction3_sqrt, 
          float lr, float decay, float eps, bool no_prox, float clip_global_grad_norm){
    // Get tensor size
    int total_size = p.numel();
    AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p),
              "parameter tensor is too large to be indexed with int32");
    
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (g.scalar_type() == at::ScalarType::Half) {
        const int block_dim = 1024;
        int grid_dim = ((total_size + block_dim - 1) / block_dim);
        const dim3 blocks(grid_dim);
        // all other values should be fp32 for half gradients
        AT_ASSERTM(p.scalar_type() == at::ScalarType::Float,
                  "expected parameter to be of float type");
        // dispatch is done on the gradient type
        using namespace at;  // prevents "toString is undefined" errors
        DISPATCH_FLOAT_AND_HALF(
            g.scalar_type(), 0, "adan_cuda_kernel",
            using accscalar_t = at::acc_type<scalar_t_0, true>;
            adan_cuda_kernel<accscalar_t, scalar_t_0>
            <<<blocks, block_dim, 0, stream>>>(
                p.data_ptr<accscalar_t>(),
                p_copy.numel() ? p_copy.data_ptr<scalar_t_0>() : NULL,
                g.data_ptr<scalar_t_0>(), exp_avg.data_ptr<accscalar_t>(), exp_avg_sq.data_ptr<accscalar_t>(),exp_avg_diff.data_ptr<accscalar_t>(), 
                neg_grad.data_ptr<scalar_t_0>(), 
                beta1, beta2, beta3, bias_correction1, bias_correction2, bias_correction3_sqrt, 
                lr, decay, eps, no_prox, clip_global_grad_norm, total_size
                );
            );
    } else {
        using namespace at;
        const int block_dim = 1024;
        int grid_dim = ((total_size + block_dim - 1) / block_dim) >> 2;
        if (grid_dim == 0) grid_dim = 1;
        const dim3 blocks(grid_dim);
        DISPATCH_DOUBLE_AND_FLOAT(
            g.scalar_type(), 0, "adan_cuda_kernel",
            adan_cuda_kernel<scalar_t_0, scalar_t_0>
            <<<blocks, block_dim, 0, stream>>>(
                p.data_ptr<scalar_t_0>(),
                NULL,
                g.data_ptr<scalar_t_0>(), exp_avg.data_ptr<scalar_t_0>(), exp_avg_sq.data_ptr<scalar_t_0>(),exp_avg_diff.data_ptr<scalar_t_0>(), 
                neg_grad.data_ptr<scalar_t_0>(), 
                beta1, beta2, beta3, bias_correction1, bias_correction2, bias_correction3_sqrt, 
                lr, decay, eps, no_prox, clip_global_grad_norm, total_size
            );
        );
    }
    AT_CUDA_CHECK(hipGetLastError());
}

