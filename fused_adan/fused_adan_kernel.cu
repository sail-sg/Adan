#include "hip/hip_runtime.h"
/* Copyright 2021 The LightSeq Team
   Copyright NVIDIA/apex
   Copyright AlexwellChen
   This kernel is adapted from NVIDIA/apex and LightSeq Team
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cmath>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/Exceptions.h>
#include "include/type_shim.h"
#include "include/fused_adan_kernel.cuh"


template <typename T, typename GRAD_T>
__global__ void adan_cuda_kernel(
    T* __restrict__ p,
    GRAD_T* __restrict__ p_copy,  // For mixed precision training, pass NULL if
                                  // not needed
    GRAD_T* __restrict__ g, T* __restrict__ exp_avg, T* __restrict__ exp_avg_sq, T* __restrict__ exp_avg_diff,
    const GRAD_T* __restrict__ neg_grad, const float b1, const float b2, const float b3, 
    const float bias_correction1, const float bias_correction2, const float bias_correction3_sqrt,
    const float lr, const float decay, const float eps, const bool no_prox, const float clip_global_grad_norm, const size_t total_size
    ){
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_id >= total_size) return;

    g[global_id] *= clip_global_grad_norm;

    GRAD_T diff, update;

    diff = g[global_id] + neg_grad[global_id];
    update = g[global_id] + b2 * diff;

    exp_avg[global_id] = b1 * exp_avg[global_id] + (1 - b1) * g[global_id];

    exp_avg_diff[global_id] = b2 * exp_avg_diff[global_id] + (1 - b2) * diff;

    exp_avg_sq[global_id] = b3 * exp_avg_sq[global_id] + (1 - b3) * update * update;

    float denom, step_size_diff, step_size;
    denom = sqrtf(exp_avg_sq[global_id]) / bias_correction3_sqrt + eps;
    step_size_diff = lr * b2 / bias_correction2;
    step_size = lr / bias_correction1;

    if (no_prox){
        p[global_id] = p[global_id] * (1 - lr * decay)
            - step_size * exp_avg[global_id] / denom
            - step_size_diff * exp_avg_diff[global_id] / denom;
    }else{
        p[global_id] = p[global_id] - step_size * exp_avg[global_id] / denom
            - step_size_diff * exp_avg_diff[global_id] / denom;
        p[global_id] = p[global_id] / (1 + lr * decay);
    } 
    if (p_copy != NULL) p_copy[global_id] = (GRAD_T)p[global_id];
}

template <>
__global__ void adan_cuda_kernel<float, float>(
    float* __restrict__ p,
    float* __restrict__ p_copy,  // For mixed precision training, pass NULL if
                                  // not needed
    float* __restrict__ g, float* __restrict__ exp_avg, float* __restrict__ exp_avg_sq, float* __restrict__ exp_avg_diff,
    const float* __restrict__ neg_grad, const float b1, const float b2, const float b3, 
    const float bias_correction1, const float bias_correction2, const float bias_correction3_sqrt,
    const float lr, const float decay, const float eps, const bool no_prox, const float clip_global_grad_norm, const size_t total_size){

        int global_id = blockIdx.x * blockDim.x + threadIdx.x;

        if (global_id * 4 >= total_size) return;

        float4* p4_ptr = reinterpret_cast<float4*>(p);
        float4* g4_ptr = reinterpret_cast<float4*>(g);
        const float4* neg_grad4_ptr = reinterpret_cast<const float4*>(neg_grad);
        float4* exp_avg4_ptr = reinterpret_cast<float4*>(exp_avg);
        float4* exp_avg_sq4_ptr = reinterpret_cast<float4*>(exp_avg_sq);
        float4* exp_avg_diff4_ptr = reinterpret_cast<float4*>(exp_avg_diff);
        
        float4 p4 = p4_ptr[global_id];
        float4 g4 = g4_ptr[global_id];
        const float4 neg_grad4 = neg_grad4_ptr[global_id];
        float4 exp_avg4 = exp_avg4_ptr[global_id];
        float4 exp_avg_sq4 = exp_avg_sq4_ptr[global_id];
        float4 exp_avg_diff4 = exp_avg_diff4_ptr[global_id];

        float4 new_p4;
        float4 new_exp_avg4;
        float4 new_exp_avg_sq4;
        float4 new_exp_avg_diff4;

        g4.x *= clip_global_grad_norm;
        g4.y *= clip_global_grad_norm;
        g4.z *= clip_global_grad_norm;
        g4.w *= clip_global_grad_norm;

        float diff1 = g4.x + neg_grad4.x;
        float diff2 = g4.y + neg_grad4.y;
        float diff3 = g4.z + neg_grad4.z;
        float diff4 = g4.w + neg_grad4.w;

        float update1 = g4.x + b2 * diff1;
        float update2 = g4.y + b2 * diff2;
        float update3 = g4.z + b2 * diff3;
        float update4 = g4.w + b2 * diff4;

        new_exp_avg4.x = b1 * exp_avg4.x + (1 - b1) * g4.x;
        new_exp_avg4.y = b1 * exp_avg4.y + (1 - b1) * g4.y;
        new_exp_avg4.z = b1 * exp_avg4.z + (1 - b1) * g4.z;
        new_exp_avg4.w = b1 * exp_avg4.w + (1 - b1) * g4.w;

        new_exp_avg_sq4.x = b3 * exp_avg_sq4.x + (1 - b3) * update1 * update1;
        new_exp_avg_sq4.y = b3 * exp_avg_sq4.y + (1 - b3) * update2 * update2;
        new_exp_avg_sq4.z = b3 * exp_avg_sq4.z + (1 - b3) * update3 * update3;
        new_exp_avg_sq4.w = b3 * exp_avg_sq4.w + (1 - b3) * update4 * update4;

        new_exp_avg_diff4.x = b2 * exp_avg_diff4.x + (1 - b2) * diff1;
        new_exp_avg_diff4.y = b2 * exp_avg_diff4.y + (1 - b2) * diff2;
        new_exp_avg_diff4.z = b2 * exp_avg_diff4.z + (1 - b2) * diff3;
        new_exp_avg_diff4.w = b2 * exp_avg_diff4.w + (1 - b2) * diff4;

        float4 denom4;
        denom4.x = sqrt(new_exp_avg_sq4.x - new_exp_avg_diff4.x * new_exp_avg_diff4.x / b2) + eps;
        denom4.y = sqrt(new_exp_avg_sq4.y - new_exp_avg_diff4.y * new_exp_avg_diff4.y / b2) + eps;
        denom4.z = sqrt(new_exp_avg_sq4.z - new_exp_avg_diff4.z * new_exp_avg_diff4.z / b2) + eps;
        denom4.w = sqrt(new_exp_avg_sq4.w - new_exp_avg_diff4.w * new_exp_avg_diff4.w / b2) + eps;
        
        float step_size_diff, step_size;
        step_size_diff = lr * b2 / bias_correction2;
        step_size = lr / bias_correction1;

        if (no_prox){
            new_p4.x = p4.x * (1 - lr * decay) - step_size * new_exp_avg4.x / denom4.x - step_size_diff * new_exp_avg_diff4.x / denom4.x;
            new_p4.y = p4.y * (1 - lr * decay) - step_size * new_exp_avg4.y / denom4.y - step_size_diff * new_exp_avg_diff4.y / denom4.y;
            new_p4.z = p4.z * (1 - lr * decay) - step_size * new_exp_avg4.z / denom4.z - step_size_diff * new_exp_avg_diff4.z / denom4.z;
            new_p4.w = p4.w * (1 - lr * decay) - step_size * new_exp_avg4.w / denom4.w - step_size_diff * new_exp_avg_diff4.w / denom4.w;
        }else{
            new_p4.x = (p4.x - step_size * new_exp_avg4.x / denom4.x - step_size_diff * new_exp_avg_diff4.x / denom4.x) / (1 + lr * decay);
            new_p4.y = (p4.y - step_size * new_exp_avg4.y / denom4.y - step_size_diff * new_exp_avg_diff4.y / denom4.y) / (1 + lr * decay);
            new_p4.z = (p4.z - step_size * new_exp_avg4.z / denom4.z - step_size_diff * new_exp_avg_diff4.z / denom4.z) / (1 + lr * decay);
            new_p4.w = (p4.w - step_size * new_exp_avg4.w / denom4.w - step_size_diff * new_exp_avg_diff4.w / denom4.w) / (1 + lr * decay);
        }   
        g4_ptr[global_id] = g4;
        p4_ptr[global_id] = new_p4;
        exp_avg4_ptr[global_id] = new_exp_avg4;
        exp_avg_sq4_ptr[global_id] = new_exp_avg_sq4;
        exp_avg_diff4_ptr[global_id] = new_exp_avg_diff4;
}

void fused_adan_cuda(at::Tensor& p, at::Tensor& p_copy, at::Tensor& g, at::Tensor& exp_avg, 
          at::Tensor& exp_avg_sq, at::Tensor& exp_avg_diff,
          at::Tensor& neg_grad, float beta1, float beta2, float beta3, 
          float bias_correction1, float bias_correction2, float bias_correction3_sqrt, 
          float lr, float decay, float eps, bool no_prox, float clip_global_grad_norm){
    // Get tensor size
    int total_size = p.numel();
    AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p),
              "parameter tensor is too large to be indexed with int32");
    
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (g.scalar_type() == at::ScalarType::Half) {
        const int block_dim = 1024;
        int grid_dim = ((total_size + block_dim - 1) / block_dim);
        const dim3 blocks(grid_dim);
        // all other values should be fp32 for half gradients
        AT_ASSERTM(p.scalar_type() == at::ScalarType::Float,
                  "expected parameter to be of float type");
        // dispatch is done on the gradient type
        using namespace at;  // prevents "toString is undefined" errors
        DISPATCH_FLOAT_AND_HALF(
            g.scalar_type(), 0, "adan_cuda_kernel",
            using accscalar_t = at::acc_type<scalar_t_0, true>;
            adan_cuda_kernel<accscalar_t, scalar_t_0>
            <<<blocks, block_dim, 0, stream>>>(
                p.data_ptr<accscalar_t>(),
                p_copy.numel() ? p_copy.data_ptr<scalar_t_0>() : NULL,
                g.data_ptr<scalar_t_0>(), exp_avg.data_ptr<accscalar_t>(), exp_avg_sq.data_ptr<accscalar_t>(),exp_avg_diff.data_ptr<accscalar_t>(), 
                neg_grad.data_ptr<scalar_t_0>(), 
                beta1, beta2, beta3, bias_correction1, bias_correction2, bias_correction3_sqrt, 
                lr, decay, eps, no_prox, clip_global_grad_norm, total_size
                );
            );
    } else {
        using namespace at;
        const int block_dim = 1024;
        int grid_dim = ((total_size + block_dim - 1) / block_dim) >> 2;
        if (grid_dim == 0) grid_dim = 1;
        const dim3 blocks(grid_dim);
        DISPATCH_DOUBLE_AND_FLOAT(
            g.scalar_type(), 0, "adan_cuda_kernel",
            adan_cuda_kernel<scalar_t_0, scalar_t_0>
            <<<blocks, block_dim, 0, stream>>>(
                p.data_ptr<scalar_t_0>(),
                NULL,
                g.data_ptr<scalar_t_0>(), exp_avg.data_ptr<scalar_t_0>(), exp_avg_sq.data_ptr<scalar_t_0>(),exp_avg_diff.data_ptr<scalar_t_0>(), 
                neg_grad.data_ptr<scalar_t_0>(), 
                beta1, beta2, beta3, bias_correction1, bias_correction2, bias_correction3_sqrt, 
                lr, decay, eps, no_prox, clip_global_grad_norm, total_size
            );
        );
    }
    AT_CUDA_CHECK(hipGetLastError());
}

