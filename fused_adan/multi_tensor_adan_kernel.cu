#include "hip/hip_runtime.h"
/* Copyright NVIDIA/apex
   Copyright AlexwellChen
   This kernel is adapted from NVIDIA/apex.
*/
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "include/type_shim.h" // Used for DISPATCH
#include "include/multi_tensor_apply.cuh" 
#include "include/fused_adan_kernel.cuh"

#define BLOCK_SIZE 512
#define ILP 4

using MATH_T = float;

template<typename T>
struct AdanFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem, // Not used here, but needed to keep same
    TensorListMetadata<6>& tl, // [0]:p, [1]:g, [2]:exp_avg, [3]:exp_avg_sq, [4]:exp_avg_diff, [5]:neg_grad
    const float beta1,
    const float beta2,
    const float beta3,
    const float bias_correction1,
    const float bias_correction2,
    const float bias_correction3_sqrt,
    const float lr,
    const float decay,
    const float epsilon,
    const bool no_prox,
    const float clip_global_grad_norm
    )
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T* p = (T*)tl.addresses[0][tensor_loc];
    p += chunk_idx*chunk_size;

    T* g = (T*)tl.addresses[1][tensor_loc];
    g += chunk_idx*chunk_size;

    T* exp_avg = (T*)tl.addresses[2][tensor_loc];
    exp_avg += chunk_idx*chunk_size;

    T* exp_avg_sq = (T*)tl.addresses[3][tensor_loc];
    exp_avg_sq += chunk_idx*chunk_size;

    T* exp_avg_diff = (T*)tl.addresses[4][tensor_loc];
    exp_avg_diff += chunk_idx*chunk_size;

    T* neg_grad = (T*)tl.addresses[5][tensor_loc];
    neg_grad += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for(int i_start = 0;
            i_start < n && i_start < chunk_size;
            i_start += blockDim.x*ILP)
    {
      MATH_T r_p[ILP];
      MATH_T r_g[ILP];
      MATH_T r_exp_avg[ILP];
      MATH_T r_exp_avg_sq[ILP];
      MATH_T r_exp_avg_diff[ILP];
      MATH_T r_neg_grad[ILP];
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          r_p[ii] = p[i];
          r_g[ii] = g[i];
          r_exp_avg[ii] = exp_avg[i];
          r_exp_avg_sq[ii] = exp_avg_sq[i];
          r_exp_avg_diff[ii] = exp_avg_diff[i];
          r_neg_grad[ii] = neg_grad[i];
        } else {
          r_p[ii] = MATH_T(0);
          r_g[ii] = MATH_T(0);
          r_exp_avg[ii] = MATH_T(0);
          r_exp_avg_sq[ii] = MATH_T(0);
          r_exp_avg_diff[ii] = MATH_T(0);
          r_neg_grad[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        r_g[ii] *= clip_global_grad_norm; //scaled_grad
        MATH_T diff, update;
        diff = r_g[ii] + r_neg_grad[ii];
        update = r_g[ii] + beta2 * diff;

        r_exp_avg[ii] = beta1 * r_exp_avg[ii] + (1 - beta1) * r_g[ii];
        r_exp_avg_diff[ii] = beta2 * r_exp_avg_diff[ii] + (1 - beta2) * diff;
        r_exp_avg_sq[ii] = beta3 * r_exp_avg_sq[ii] + (1 - beta3) * update * update;

        MATH_T denom;
        denom = sqrtf(r_exp_avg_sq[ii]) / bias_correction3_sqrt + epsilon;
        update = (r_exp_avg[ii] / bias_correction1 + beta2 * r_exp_avg_diff[ii] / bias_correction2) / denom;
        
        if(no_prox){
          r_p[ii] = r_p[ii] * (1 - lr * decay) + update * (-lr);
        } else {
          r_p[ii] = r_p[ii] + update * (-lr) / (1 + lr * decay);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          p[i] = r_p[ii];
          exp_avg[i] = r_exp_avg[ii];
          exp_avg_sq[i] = r_exp_avg_sq[ii];
          exp_avg_diff[i] = r_exp_avg_diff[ii];
        }
      }
    }
  }
};

void multi_tensor_adan_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  const float beta1,
  const float beta2,
  const float beta3,
  const float bias_correction1,
  const float bias_correction2,
  const float bias_correction3_sqrt,
  const float lr,
  const float decay,
  const float epsilon,
  const bool no_prox,
  const float clip_global_grad_norm)
{
  using namespace at;

  // Assume single type across p,g,m1,m2 now
  DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
    tensor_lists[0][0].scalar_type(), 0, "adan",
    multi_tensor_apply<6>(
      BLOCK_SIZE,
      chunk_size,
      noop_flag,
      tensor_lists,
      AdanFunctor<scalar_t_0>(),
      beta1,
      beta2,
      beta3,
      bias_correction1,
      bias_correction2,
      bias_correction3_sqrt,
      lr,
      decay,
      epsilon,
      no_prox,
      clip_global_grad_norm
      ); )

  AT_CUDA_CHECK(hipGetLastError());

}
